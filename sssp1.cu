#include<stdio.h>
#include<hip/hip_runtime.h>
#include<utility>

#define pii std::pair<int,int>
#define piii std::pair<int,std::pair<int,int>>
#define mp std::make_pair
#define f first
#define s second 

//declaration of class object - graph

class graph
{
public:
    
    int v,e; //number of vertices and edges
    int *distance; //array that stores distance of each vertex from source
    piii *edgelist; //list of edges in the form of <weight,vertex1,vertex2> tuples  

};

//Kernel that initializes distance array of graph

__global__ void init_distance_kernel(graph *g, int start)
{
    unsigned int id = blockIdx.x*blockDim.x + threadIdx.x; //id of thread

    //checking if vertex is the source vertex or not, and initializing distance value correspondingly
    if(id == start)
    {
        g->distance[id] = 0;
    }

    else
    {
        g->distance[id] = INT_MAX>>2;
    }

}

//Kernel invoked in sssp routine

__global__ void sssp_kernel(graph *g)
{
    unsigned int id = blockIdx.x*blockDim.x + threadIdx.x; //id of thread

    //storing corresponding weight, vertices and distance values into thread's local memory 
    int w = g->edgelist[id].f;
    int v1 = g->edgelist[id].s.f;
    int v2 = g->edgelist[id].s.s;
    int d1 = g->distance[v1];
    int d2 = g->distance[v2];

    __syncthreads(); //to create a barrier and sync all threads
    
    /*Implementing relax routine of bellman ford algorithm
     *AtomicMin is used to resolve read-write conflicts during interleaving
     */
    atomicMin(&(g->distance[v1]),(d2+w));
    atomicMin(&(g->distance[v2]),(d1+w));
    
}

//host function for simple_bfs routine that invokes sssp_kernel iteratively 

void sssp(graph *cpu_g, graph *gpu_g)
{
    int i = (cpu_g->v - 1);
    while(i--)
    {
        sssp_kernel<<<1,cpu_g->e>>>(gpu_g);
    }

}


//host function that reads graph data from command line

void readgraph(piii *c_edgelist, int nv, int ne, int argc, char **argv)
{
    if(argc <= 4 || argc%3 != 1)
    {
        printf("Enter valid arguments in command line\n");
        exit(0);
    }

    else
    {
        int i,j;
        for(i=0, j=4;j<argc-2;i++,j+=3)
        {
            int wt = atoi(argv[j]);
            int e1 = atoi(argv[j+1]);
            int e2 = atoi(argv[j+2]);
            c_edgelist[i] = mp(wt,mp(e1,e2));
        }
    }
    
}

//host function to print the distance of each vertex

void printgraph(graph *cpu_g)
{
    for(int i=0;i<cpu_g->v;i++)
    {
        printf("The distance of vertex %d is %d\n",i,cpu_g->distance[i]);
    }
}

//main function

int main(int argc, char **argv)
{
    //declaration of variables to store graph data on host and device
    graph *cpu_g,*gpu_g;
    int *c_distance,*g_distance;
    piii *c_edgelist,*g_edgelist;

    //asking user to run with inputs in command line if no inputs are given
    if(argc == 1)
    {
        printf("Enter arguments in command line\n");
        return 0;
    }

    int nv = atoi(argv[1]); //number of vertices
    int ne = atoi(argv[2]); //number of edges
    int start = atoi(argv[3]);  //source vertex

    //allocating host memory for data of graph in host
    c_distance = (int*)malloc(nv*sizeof(int));
    c_edgelist = (piii *)malloc(ne*sizeof(piii));

    //invoking function to read graph data from command line
    readgraph(c_edgelist,nv,ne,argc,argv);

    //allocating host memory for graph object
    cpu_g = (graph*)malloc(sizeof(graph));

    //assigning values to data members of graph object from host data
    cpu_g->v = nv;
    cpu_g->e = ne;
    cpu_g->distance = c_distance;
    cpu_g->edgelist = c_edgelist;

    //allocating device memory for graph object on GPU    
    hipMalloc((void**)&gpu_g,sizeof(graph));
    hipMalloc((void**)&g_distance,nv*sizeof( int));
    hipMalloc((void**)&g_edgelist,ne*sizeof(piii));

    //copying host data onto device
    hipMemcpy(g_distance,c_distance,nv*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(g_edgelist,c_edgelist,ne*sizeof(piii),hipMemcpyHostToDevice);
    hipMemcpy(gpu_g,cpu_g,sizeof(graph),hipMemcpyHostToDevice);

    hipMemcpy(&(gpu_g->edgelist),&g_edgelist,sizeof(piii *),hipMemcpyHostToDevice);
    hipMemcpy(&(gpu_g->distance),&g_distance,sizeof(int*),hipMemcpyHostToDevice);

    //invoking kernel to initialize the distance array of the graph 
    init_distance_kernel<<<1,nv>>>(gpu_g,start);

    //declaration of bool variables in host - used for routine to invoke bfs kernel
    /*bool *cpu_done;
    cpu_done = (bool*)malloc(sizeof(bool));
    *cpu_done = false;

    //declaration of bool variables in device for routine to invoke bfs kernel   
    bool *gpu_done;
    cudaMalloc((void**)&gpu_done,sizeof(bool));
    cudaMemcpy(gpu_done,cpu_done,sizeof(bool),cudaMemcpyHostToDevice);*/

    //routine that invokes sssp kernel from host
    sssp(cpu_g,gpu_g);
    //cudaDeviceSynchronize();

    //copying device data back into host memory
    hipMemcpy(c_distance,g_distance,nv*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(c_edgelist,g_edgelist,ne*sizeof(piii),hipMemcpyDeviceToHost);
    hipMemcpy(cpu_g,gpu_g,sizeof(graph),hipMemcpyDeviceToHost);
    
    cpu_g->edgelist = c_edgelist;
    cpu_g->distance = c_distance;

    //printing distance of vertices from host memory
    printgraph(cpu_g);
   
    return 0;

}