#include"include/graph.cuh"

//main function

int main(int argc, char **argv)
{
    //declaration of variables to store graph data on host and device
    graph *cpu_g,*gpu_g;
    int *c_distance,*g_distance;
    piii *c_edgelist,*g_edgelist;

    //asking user to run with inputs in command line if no inputs are given
    if(argc == 1)
    {
        printf("Enter arguments in command line\n");
        return 0;
    }

    int nv = atoi(argv[1]); //number of vertices
    int ne = atoi(argv[2]); //number of edges
    int start = atoi(argv[3]);  //source vertex

    //allocating host memory for data of graph in host
    c_distance = (int*)malloc(nv*sizeof(int));
    c_edgelist = (piii *)malloc(ne*sizeof(piii));

    //invoking function to read graph data from command line
    readgraph(c_edgelist,nv,ne,argc,argv);

    //allocating host memory for graph object
    cpu_g = (graph*)malloc(sizeof(graph));

    //assigning values to data members of graph object from host data
    cpu_g->v = nv;
    cpu_g->e = ne;
    cpu_g->distance = c_distance;
    cpu_g->edgelist = c_edgelist;

    //allocating device memory for graph object on GPU    
    hipMalloc((void**)&gpu_g,sizeof(graph));
    hipMalloc((void**)&g_distance,nv*sizeof( int));
    hipMalloc((void**)&g_edgelist,ne*sizeof(piii));

    //copying host data onto device
    hipMemcpy(g_distance,c_distance,nv*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(g_edgelist,c_edgelist,ne*sizeof(piii),hipMemcpyHostToDevice);
    hipMemcpy(gpu_g,cpu_g,sizeof(graph),hipMemcpyHostToDevice);

    hipMemcpy(&(gpu_g->edgelist),&g_edgelist,sizeof(piii *),hipMemcpyHostToDevice);
    hipMemcpy(&(gpu_g->distance),&g_distance,sizeof(int*),hipMemcpyHostToDevice);

    //invoking kernel to initialize the distance array of the graph 
    init_distance_kernel<<<1,nv>>>(gpu_g,start);

    //routine that invokes sssp kernel from host
    sssp(cpu_g,gpu_g);

    //copying device data back into host memory
    hipMemcpy(c_distance,g_distance,nv*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(c_edgelist,g_edgelist,ne*sizeof(piii),hipMemcpyDeviceToHost);
    hipMemcpy(cpu_g,gpu_g,sizeof(graph),hipMemcpyDeviceToHost);
    
    cpu_g->edgelist = c_edgelist;
    cpu_g->distance = c_distance;

    printf("From source vertex %d :\n",start);

    //printing distance of vertices from host memory
    printgraph(cpu_g);
   
    return 0;

}