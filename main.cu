#include "hip/hip_runtime.h"
#include"include/graph.cuh"
#include<bits/stdc++.h>

//main function

int main(int argc, char **argv)
{
    //declaration of variables to store graph data on host and device
    graph *cpu_g,*gpu_g;
    int *c_distance,*g_distance;
    piii *c_edgelist,*g_edgelist;

    //asking user to run with inputs in command line if no inputs are given
    if(argc == 1)
    {
        printf("Enter arguments in command line\n");
        return 0;
    }

    int nv = atoi(argv[1]); //number of vertices
    int ne = atoi(argv[2]); //number of edges
    int start = atoi(argv[3]);  //source vertex

    //allocating host memory for data of graph in host
    c_distance = (int*)malloc(nv*sizeof(int));
    c_edgelist = (piii *)malloc(ne*sizeof(piii));

    //invoking function to read graph data from command line
    readgraph(c_edgelist,nv,ne,argc,argv);

    //allocating host memory for graph object
    cpu_g = (graph*)malloc(sizeof(graph));

    //assigning values to data members of graph object from host data
    cpu_g->v = nv;
    cpu_g->e = ne;
    cpu_g->distance = c_distance;
    cpu_g->edgelist = c_edgelist;

    //allocating device memory for graph object on GPU    
    hipMalloc((void**)&gpu_g,sizeof(graph));
    hipMalloc((void**)&g_distance,nv*sizeof( int));
    hipMalloc((void**)&g_edgelist,ne*sizeof(piii));

    //copying host data onto device
    hipMemcpy(g_distance,c_distance,nv*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(g_edgelist,c_edgelist,ne*sizeof(piii),hipMemcpyHostToDevice);
    hipMemcpy(gpu_g,cpu_g,sizeof(graph),hipMemcpyHostToDevice);

    hipMemcpy(&(gpu_g->edgelist),&g_edgelist,sizeof(piii *),hipMemcpyHostToDevice);
    hipMemcpy(&(gpu_g->distance),&g_distance,sizeof(int*),hipMemcpyHostToDevice);

    std::clock_t start_time,end_time;
    start_time = std::clock();

    //invoking kernel to initialize the distance array of the graph 
    init_distance_kernel<<<nblocks,threads_per_block>>>(gpu_g,start);

    //routine that invokes sssp kernel from host
    sssp(cpu_g,gpu_g);

    end_time = std::clock();

    //copying device data back into host memory
    hipMemcpy(c_distance,g_distance,nv*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(c_edgelist,g_edgelist,ne*sizeof(piii),hipMemcpyDeviceToHost);
    hipMemcpy(cpu_g,gpu_g,sizeof(graph),hipMemcpyDeviceToHost);
    
    cpu_g->edgelist = c_edgelist;
    cpu_g->distance = c_distance;

    double time_taken = double(end_time - start_time) / double(CLOCKS_PER_SEC); 
    
    FILE *g = fopen("time.txt","a");
    fprintf(g,"%lf\n",time_taken); 

    printf("Exec time written in time.txt\n");


    //printf("From source vertex %d :\n",start); 

    //printing distance of vertices from host memory
    //printgraph(cpu_g);

    //freeing memory allocated on GPU
    hipFree(g_distance);
    hipFree(g_edgelist);
    hipFree(gpu_g);

    //freeing memory allocated on CPU
    free(c_distance);
    free(c_edgelist);
    free(cpu_g);
 
   
    return 0;

}