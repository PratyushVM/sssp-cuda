#include "hip/hip_runtime.h"
#include"../include/graph.cuh"

//Kernel invoked in parallel bellman ford sssp routine

__global__ void sssp_kernel(graph *g)
{
    unsigned int id = blockIdx.x*blockDim.x + threadIdx.x; //id of thread

    //storing corresponding weight, vertices and distance values into thread's local memory 
    int w = g->edgelist[id].f; //weight of edge
    int v1 = g->edgelist[id].s.f; //end 1 of edge
    int v2 = g->edgelist[id].s.s; //end 2 of edge
    int d1 = g->distance[v1]; //distance value of end 1 at that iteration
    int d2 = g->distance[v2]; //distance value of end 2 at that iteration

    __syncthreads(); //to create a barrier and sync all threads
    
    /*Implementing relax routine of bellman ford algorithm
     *AtomicMin is used to resolve read-write conflicts during interleaving
     */
    atomicMin(&(g->distance[v1]),(d2+w));
    atomicMin(&(g->distance[v2]),(d1+w));
    
}