#include "hip/hip_runtime.h"
#include"../include/graph.cuh"

//Kernel invoked in parallel bellman ford sssp routine

__global__ void sssp_kernel(graph *g)
{
    if(threadIdx.x < g->e)
    {    
        unsigned int id = blockIdx.x*blockDim.x + threadIdx.x; //id of thread

        //storing corresponding weight, vertices and distance values into thread's local memory 
        int w = g->edgelist[id].first; //weight of edge
        int v1 = g->edgelist[id].second.first; //end 1 of edge
        int v2 = g->edgelist[id].second.second; //end 2 of edge
        int d1 = g->distance[v1]; //distance value of end 1 at that iteration
        int d2 = g->distance[v2]; //distance value of end 2 at that iteration

        //__syncthreads(); //to sync all threads - so that all threads take only the value of the previous iteration
                         //preserves invariant of the algorithm 
        
        /*Implementing relax routine of bellman ford algorithm
        *AtomicMin is used to resolve read-write conflicts during interleaving
        */
        
        atomicMin(&(g->distance[v1]),(d2+w));
        atomicMin(&(g->distance[v2]),(d1+w)); 
    }
        
}