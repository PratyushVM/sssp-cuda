#include "hip/hip_runtime.h"
#include"../include/graph.cuh"

//Kernel that initializes distance array of graph

__global__ void init_distance_kernel(graph *g, int start)
{
    unsigned int id = blockIdx.x*blockDim.x + threadIdx.x; //id of thread

    //checking if vertex is the source vertex or not, and initializing distance value correspondingly
    if(id == start)
    {
        g->distance[id] = 0;
    }

    else
    {
        g->distance[id] = INT_MAX>>2;
    }

}